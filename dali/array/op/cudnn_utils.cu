#include "cudnn_utils.h"

#include <atomic>

///////////////////////////////////////////////////////////////////////////////
//                              UTILS                                        //
///////////////////////////////////////////////////////////////////////////////

#define CUDNN_CHECK_RESULT(status, message) \
        ASSERT2(status == HIPDNN_STATUS_SUCCESS, \
            utils::MS() << message << ", cudnn error: " << hipdnnGetErrorString(status))

std::string cudnnGetTensorFormatString(hipdnnTensorFormat_t tf) {
    if (tf == HIPDNN_TENSOR_NCHW) {
        return "NCHW";
    } else if (tf == HIPDNN_TENSOR_NHWC) {
        return "NHWC";
    } else {
        return "unknown";
    }
}

std::string cudnnGetDateTypeString(hipdnnDataType_t dt) {
    if (dt == HIPDNN_DATA_HALF) {
        return "float16";
    } else if (dt == HIPDNN_DATA_FLOAT) {
        return "float32";
    } else if (dt == HIPDNN_DATA_DOUBLE) {
        return "float64";
    } else {
        return "unknown";
    }
}


template<typename T>
struct TensorWrapperApi {
};

template<>
struct TensorWrapperApi<hipdnnTensorDescriptor_t> {
    static void create(hipdnnTensorDescriptor_t* doodle) {
        auto result = hipdnnCreateTensorDescriptor(doodle);
        CUDNN_CHECK_RESULT(result, "when creating tensor descriptor");
    }

    static void destroy(hipdnnTensorDescriptor_t doodle) {
        auto result = hipdnnDestroyTensorDescriptor(doodle);
        CUDNN_CHECK_RESULT(result, "when destroying tensor descriptor");

    }

    static void set(hipdnnTensorDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int n,
                    int c,
                    int h,
                    int w) {
        auto result = hipdnnSetTensor4dDescriptor(
            desc,
            tensor_format,
            dtype,
            n,
            c,
            h,
            w);

        CUDNN_CHECK_RESULT(result, "when setting tensor descriptor with "
                << "shape = [n=" << n << ",c=" << c << ",h=" << h << ",w=" << w << "], "
                << "data format = " << cudnnGetTensorFormatString(tensor_format) << ", "
                << "dtype = " << cudnnGetDateTypeString(dtype));
    }
};

template<>
struct TensorWrapperApi<hipdnnFilterDescriptor_t> {
    static void create(hipdnnFilterDescriptor_t* doodle) {
        auto result = hipdnnCreateFilterDescriptor(doodle);
        CUDNN_CHECK_RESULT(result, "when creating filter descriptor");

    }

    static void destroy(hipdnnFilterDescriptor_t doodle) {
        auto result = hipdnnDestroyFilterDescriptor(doodle);
        CUDNN_CHECK_RESULT(result, "when destroying filter descriptor");
    }

    static void set(hipdnnFilterDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int n,
                    int c,
                    int h,
                    int w) {
        auto result = hipdnnSetFilter4dDescriptor(
            desc,
            dtype,
            tensor_format,
            n,
            c,
            h,
            w);

        CUDNN_CHECK_RESULT(result, "when setting filter descriptor with "
                << "shape = [n=" << n << ",c=" << c << ",h=" << h << ",w=" << w << "], "
                << "data format = " << cudnnGetTensorFormatString(tensor_format) << ", "
                << "dtype = " << cudnnGetDateTypeString(dtype));
    }
};

static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);

// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

namespace cudnn {
    ///////////////////////////////////////////////////////////////////////////
    //                              wrappers                                 //
    ///////////////////////////////////////////////////////////////////////////


    namespace wrapper {
        template<typename Descriptor>
        template<typename T, int devT>
        BaseTensor<Descriptor>::BaseTensor(
                TypedArray<devT,T> tensor,
                std::string data_format,
                memory::AM access_mode) {
            ASSERT2(devT == memory::DEVICE_T_GPU,
                    "cudnn Tensor/Filters wrapper must be "
                    "constructed from GPU TypedArray.");
            hipdnnTensorFormat_t data_format_cudnn;
            if (data_format == "NCHW") {
                data_format_cudnn = HIPDNN_TENSOR_NCHW;
            } else if (data_format == "NHWC") {
                data_format_cudnn = HIPDNN_TENSOR_NHWC;
            } else {
                ASSERT2(false, "unsupported data format");
            }
            hipdnnDataType_t cudnn_dtype;
            if (template_to_dtype<T>() == DTYPE_FLOAT) {
                cudnn_dtype = HIPDNN_DATA_FLOAT;
            } else if (template_to_dtype<T>() == DTYPE_DOUBLE) {
                cudnn_dtype = HIPDNN_DATA_DOUBLE;
            } else {
                ASSERT2(false, "unsupported dtype");
            }
            int shape0, shape1, shape2, shape3;
            if (tensor.array.ndim() == 1) {
                if (data_format == "NCHW") {
                    shape0 = shape2 = shape3 = 1;
                    shape1 = tensor.array.shape()[0];
                } else if (data_format == "NHWC") {
                    shape0 = shape1 = shape2 = 1;
                    shape3 = tensor.array.shape()[0];
                }
            } else if (tensor.array.ndim() == 4) {
                shape0 = tensor.array.shape()[0];
                shape1 = tensor.array.shape()[1];
                shape2 = tensor.array.shape()[2];
                shape3 = tensor.array.shape()[3];
            } else {
                ASSERT2(false, "cudnn::wrapper::Tensor can only support 1D and 4D tensors.");
            }

            int n,c,h,w;
            if (data_format == "NCHW") {
                n = shape0; c = shape1; h = shape2; w = shape3;
            } else {
                n = shape0; c = shape3; h = shape1; w = shape2;
            }

            TensorWrapperApi<Descriptor>::create(&description);
            TensorWrapperApi<Descriptor>::set(
                description,
                data_format_cudnn,
                cudnn_dtype,
                n,
                c,
                h,
                w
            );
            // TODO(szymon): add striding support and assert maybe???
            data = tensor.ptr(access_mode);
        }

        template<typename Descriptor>
        BaseTensor<Descriptor>::~BaseTensor() {
            TensorWrapperApi<Descriptor>::destroy(description);
        }

        template class BaseTensor<hipdnnTensorDescriptor_t>;
        template class BaseTensor<hipdnnFilterDescriptor_t>;

        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,int>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,double>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,int>, std::string, memory::AM);

        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,int>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,double>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_CPU,int>, std::string, memory::AM);


        Convolution::Convolution(int padding_h, int padding_w,
                                 int stride_h, int stride_w) {
            auto result = hipdnnCreateConvolutionDescriptor(&description);
            CUDNN_CHECK_RESULT(result, "when creating convolution descriptor");
            result = hipdnnSetConvolution2dDescriptor(
                description,
                /*pad_h=*/   padding_h,
                /*pad_w=*/   padding_w,
                /*u=*/       stride_h,
                /*v=*/       stride_w,
                /*upscalex=*/1,
                /*upscaley=*/1,
                HIPDNN_CROSS_CORRELATION // Theano issue author claims its twice as fast:
                                        // https://github.com/Theano/Theano/issues/3632
            );


            CUDNN_CHECK_RESULT(result, "when setting convolution descriptor with "
                << "padding_h = "  << padding_h << ", padding_w = " << padding_w
                << ", stride_h = " << stride_h  << ", stride_w = "  << stride_w);
        }

        Convolution::~Convolution() {
            auto result = hipdnnDestroyConvolutionDescriptor(description);
            CUDNN_CHECK_RESULT(result, "when destroying convolution descriptor");
        }

        Pooling::Pooling(int window_h,  int window_w,
                         int padding_h, int padding_w,
                         int stride_h,  int stride_w,
                         POOLING_T pooling_mode) {
            auto result = hipdnnCreatePoolingDescriptor(&description);
            CUDNN_CHECK_RESULT(result, "when creating pooling descriptor");

            hipdnnPoolingMode_t cudnn_pooling_mode;
            if (pooling_mode == POOLING_T_MAX) {
                cudnn_pooling_mode = HIPDNN_POOLING_MAX;
            } else if (pooling_mode == POOLING_T_AVG) {
                // Following what TensorFlow does:
                //   https://github.com/tensorflow/tensorflow/blob/
                //   6431560b7ec3565154cb9cdc9c827db78ccfebe7/
                //   tensorflow/stream_executor/cuda/cuda_dnn.cc
                cudnn_pooling_mode =
                        HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
            } else {
                ASSERT2(false, utils::MS() << "unknown POOLING_T ("
                                           << pooling_mode << ").");
            }

            result = hipdnnSetPooling2dDescriptor(
                description,
                cudnn_pooling_mode,
                HIPDNN_PROPAGATE_NAN,
                /*windowHeight=*/ window_h,
                /*windowWidth=*/  window_w,
                /*pad_h=*/        padding_h,
                /*pad_w=*/        padding_w,
                /*stride_h=*/     stride_h,
                /*stride_w=*/     stride_w
            );
            CUDNN_CHECK_RESULT(result, "when setting Pooling descriptor");
        }

        Pooling::~Pooling() {
            auto result = hipdnnDestroyPoolingDescriptor(description);
            CUDNN_CHECK_RESULT(result, "when destroying Pooling descriptor");
        }


        Operator::Operator(OPERATOR_T operator_type, DType dtype) {
            switch (operator_type) {
                case OPERATOR_T_EQL:
                    alpha_d = 1.0;
                    beta_d  = 0.0;
                    break;
                case OPERATOR_T_ADD:
                    alpha_d = 1.0;
                    beta_d  = 1.0;
                    break;
                case OPERATOR_T_SUB:
                    alpha_d = -1.0;
                    beta_d  = 1.0;
                    break;
                default:
                    ASSERT2(false, "Cudnn only supports =, + and - operators");
            }

            alpha_f = alpha_d;
            beta_f  = beta_d;

            if (dtype == DTYPE_FLOAT) {
                alpha_ptr = (void*)&alpha_f;
                beta_ptr  = (void*)&beta_f;
            } else if (dtype == DTYPE_DOUBLE) {
                alpha_ptr = (void*)&alpha_d;
                beta_ptr  = (void*)&beta_d;
            } else {
                ASSERT2(false, "Cudnn only supports floating point types");
            }
        }
    }  // namespace wrapper


    ///////////////////////////////////////////////////////////////////////////
    //                              CONVOLUTIONS                             //
    ///////////////////////////////////////////////////////////////////////////

    void conv2d(std::shared_ptr<wrapper::Tensor>  out,
                std::shared_ptr<wrapper::Tensor>  in,
                std::shared_ptr<wrapper::Filters> filters,
                std::shared_ptr<wrapper::Convolution> conv,
                const wrapper::Operator& update_operator) {
        // TODO(szymon): automatically choose best algorithm.
        hipdnnConvolutionFwdAlgo_t algo =
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
        void* working_memory    = NULL;
        int working_memory_size = 0;

        auto result = hipdnnConvolutionForward(
            *get_handle(),
            update_operator.alpha_ptr,
            in->description,
            in->data,
            filters->description,
            filters->data,
            conv->description,
            algo,
            working_memory,
            working_memory_size,
            update_operator.beta_ptr,
            out->description,
            out->data
        );
        CUDNN_CHECK_RESULT(result, "when running hipdnnConvolutionForward");
    }

    void conv2d_bwd_input(std::shared_ptr<wrapper::Tensor>  in_dw,
                          std::shared_ptr<wrapper::Filters> filters,
                          std::shared_ptr<wrapper::Tensor>  out_dw,
                          std::shared_ptr<wrapper::Convolution> conv,
                          const wrapper::Operator& update_operator) {
        // TODO(szymon): automatically choose best algorithm.
        hipdnnConvolutionBwdDataAlgo_t algo =
                HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
        void* working_memory    = NULL;
        int working_memory_size = 0;

        auto result = hipdnnConvolutionBackwardData(
            *get_handle(),
            update_operator.alpha_ptr,
            filters->description,
            filters->data,
            out_dw->description,
            out_dw->data,
            conv->description,
            algo,
            working_memory,
            working_memory_size,
            update_operator.beta_ptr,
            in_dw->description,
            in_dw->data
        );
        CUDNN_CHECK_RESULT(result, "when computing convolution's data gradient");
    }


    void conv2d_bwd_filters(std::shared_ptr<wrapper::Filters> filters_dw,
                            std::shared_ptr<wrapper::Tensor>  input,
                            std::shared_ptr<wrapper::Tensor>  out_dw,
                            std::shared_ptr<wrapper::Convolution> conv,
                            const wrapper::Operator& update_operator) {
        // TODO(szymon): automatically choose best algorithm.
        hipdnnConvolutionBwdFilterAlgo_t algo =
                HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;
        void* working_memory    = NULL;
        int working_memory_size = 0;

        auto result = hipdnnConvolutionBackwardFilter(
            *get_handle(),
            update_operator.alpha_ptr,
            input->description,
            input->data,
            out_dw->description,
            out_dw->data,
            conv->description,
            algo,
            working_memory,
            working_memory_size,
            update_operator.beta_ptr,
            filters_dw->description,
            filters_dw->data
        );
        CUDNN_CHECK_RESULT(result, "when computing convolution's filter gradient");
    }

    void conv2d_bwd_bias(std::shared_ptr<wrapper::Tensor> bias_dw,
                         std::shared_ptr<wrapper::Tensor> out_dw,
                         const wrapper::Operator& update_operator) {
        auto result = hipdnnConvolutionBackwardBias(
            *get_handle(),
            update_operator.alpha_ptr,
            out_dw->description,
            out_dw->data,
            update_operator.beta_ptr,
            bias_dw->description,
            bias_dw->data
        );
        CUDNN_CHECK_RESULT(result, "when computing convolution bias gradient");
    }


    void pool2d(std::shared_ptr<wrapper::Tensor> out,
                std::shared_ptr<wrapper::Tensor>  in,
                std::shared_ptr<wrapper::Pooling> pooling,
                const wrapper::Operator& update_operator) {

        auto result = hipdnnPoolingForward(
            *get_handle(),
            pooling->description,
            update_operator.alpha_ptr,
            in->description,
            in->data,
            update_operator.beta_ptr,
            out->description,
            out->data
        );

        CUDNN_CHECK_RESULT(result, "when computing pooling forward");
    }

        void pool2d_bwd(std::shared_ptr<wrapper::Tensor> in_dw,
                        std::shared_ptr<wrapper::Tensor> out,
                        std::shared_ptr<wrapper::Tensor> out_dw,
                        std::shared_ptr<wrapper::Tensor> in,
                        std::shared_ptr<wrapper::Pooling> pooling,
                        const wrapper::Operator& update_operator) {
            auto result = hipdnnPoolingBackward(
                *get_handle(),
                pooling->description,
                update_operator.alpha_ptr,
                out->description,
                out->data,
                out_dw->description,
                out_dw->data,
                in->description,
                in->data,
                update_operator.beta_ptr,
                in_dw->description,
                in_dw->data
            );

            CUDNN_CHECK_RESULT(result, "when computing pooling forward");
        }

}  // namespace cudnn
