#include "dali/utils/gpu_utils.h"

namespace gpu_utils {
    void set_default_gpu(int device) {
        hipSetDevice(device);
    }

    std::string get_gpu_name(int device) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device);

        return std::string(props.name);
    }

    int num_gpus() {
        int devices;
        hipGetDeviceCount(&devices);
        return devices;
    }
}
