#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <mshadow/tensor.h>
#include "dali/math/TensorInternal.h"
#include "dali/math/ThrustSoftmax.h"
#include "dali/utils/core_utils.h"
#include "dali/math/memory_bank/MemoryBank.h"

using std::vector;


typedef float R;

template<int buffer_bits, typename R,  typename DstPlan, typename SrcPlan>
__global__ void SoftmaxKernel(DstPlan dst, SrcPlan src, mshadow::index_t num_rows, R temperature) {
    const unsigned buffer_size = 1 << buffer_bits;
    const int column     = blockIdx.y;
    const int thread_idx = threadIdx.x;
    __shared__ R buffer[buffer_size];
    // step 1: get max
    if (thread_idx < num_rows) {
        buffer[thread_idx] = src.Eval(thread_idx, column);
    }
    for (unsigned offset = buffer_size; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            const R a = src.Eval(row, column);
            buffer[thread_idx] = max(buffer[thread_idx], a);
        }
    }
    __syncthreads();
    // if number of rows is smaller than buffer,
    // fill buffer with copy of buffer[0] - this
    // makes sure reduction does not use uninitialized
    // values in the buffer and returns correct max.
    if (thread_idx >= num_rows) {
        buffer[thread_idx] = buffer[0];
    }
    __syncthreads();
    mshadow::cuda::ReduceX<mshadow::red::maximum, buffer_bits, R>(buffer, thread_idx);
    __syncthreads();
    // every thread memorizes max value in column,
    // so that we can reuse the buffer, for next
    // task
    R max_value_in_column = buffer[0];
    __syncthreads();
    // clear buffer (so that sum works out later)
    buffer[thread_idx] = 0.0f;
    __syncthreads();
    for (unsigned offset = 0; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            R p = expf((src.Eval(row, column) - max_value_in_column) / temperature);
            // add sum to buffer, so that we can later reduce it to
            // column-wise sum of exps and use as normalizer.
            buffer[thread_idx] += p;
            // save exped value to the corresponding idx in destination.
            dst.REval(row, column) = p;
        }
    }
    __syncthreads();
    // calculate normalizer by reducing partial sums
    mshadow::cuda::ReduceX<mshadow::red::sum, buffer_bits, R>(buffer, thread_idx);
    __syncthreads();
    R colwise_sum = buffer[0];

    for (unsigned offset = 0; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            dst.REval(row, column) /= colwise_sum;
        }
    }
}

template<int buffer_bits, typename R,  typename DstPlan, typename SrcPlan>
__global__ void SoftmaxKernelCached(DstPlan dst, SrcPlan src, mshadow::index_t num_rows, R temperature) {
    const unsigned buffer_size = 1 << buffer_bits;
    const int column     = blockIdx.y;
    const int thread_idx = threadIdx.x;
    extern __shared__ R buffer[];
    R* row_cache = buffer + buffer_size;

    for (unsigned offset = 0; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            row_cache[row] =  src.Eval(row, column);
        }
    }

    // step 1: get max

    if (thread_idx < num_rows) {
        buffer[thread_idx] = row_cache[thread_idx];
    }
    for (unsigned offset = buffer_size; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            const R a = row_cache[row];
            buffer[thread_idx] = max(buffer[thread_idx], a);
        }
    }
    __syncthreads();
    // if number of rows is smaller than buffer,
    // fill buffer with copy of buffer[0] - this
    // makes sure reduction does not use uninitialized
    // values in the buffer and returns correct max.
    if (thread_idx >= num_rows) {
        buffer[thread_idx] = buffer[0];
    }
    __syncthreads();
    mshadow::cuda::ReduceX<mshadow::red::maximum, buffer_bits, R>(buffer, thread_idx);
    __syncthreads();
    // every thread memorizes max value in column,
    // so that we can reuse the buffer, for next
    // task
    R max_value_in_column = buffer[0];
    __syncthreads();
    // clear buffer (so that sum works out later)
    buffer[thread_idx] = 0.0f;
    __syncthreads();
    for (unsigned offset = 0; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            const R p = expf((src.Eval(row, column) - max_value_in_column) / temperature);
            // add sum to buffer, so that we can later reduce it to
            // column-wise sum of exps and use as normalizer.
            buffer[thread_idx] += p;
            // save exped value to the corresponding idx in destination.
            row_cache[row] = p;
        }
    }
    __syncthreads();
    // calculate normalizer by reducing partial sums
    mshadow::cuda::ReduceX<mshadow::red::sum, buffer_bits, R>(buffer, thread_idx);
    __syncthreads();
    R colwise_sum = buffer[0];

    for (unsigned offset = 0; offset < num_rows; offset += buffer_size) {
        const int row = offset + thread_idx;
        if (row < num_rows) {
            dst.REval(row, column) = row_cache[row] / colwise_sum;
        }
    }
}


// Note: in a dim3 (width, height, depth)
// every uninitialized dimension defaults to 1.

// Note: <<<Dg, Db, Ns, S>>> CUDA Language Extension is explained here:
// http://docs.nvidia.com/cuda/cuda-c-programming-guide/#execution-configuration
template<typename R>
void softmax(mshadow::Tensor<mshadow::gpu, 2, R> dst,
                    const mshadow::Tensor<mshadow::gpu, 2, R> src, R temperature = 1.0) {
  const int num_threads = 1024;//mshadow::cuda::kBaseThreadNum;
  const int thread_bits = 10;//mshadow::cuda::kBaseThreadBits;

  const int MAX_SHAREDMEM_SIZE = 65536;

  dim3 tiles(1, dst.size(1));
  // block size is a matrix column
  dim3 within_tile(num_threads);
  mshadow::utils::Check(dst.shape_ == src.shape_, "Softmax: shape mismatch");
  // mshadow::cuda::CheckLaunchParam(blockGridRows, threadBlockRows, "Softmax");
  hipStream_t stream = mshadow::Stream<mshadow::gpu>::GetStream(dst.stream_);

  const int shared_mem_for_cached = (num_threads + dst.size(0)) * sizeof(R);
  if (shared_mem_for_cached < MAX_SHAREDMEM_SIZE) {
      SoftmaxKernelCached<thread_bits, R>
          <<<tiles, within_tile, shared_mem_for_cached, stream>>>
          (mshadow::expr::MakePlan(dst),
           mshadow::expr::MakePlan(src),
           dst.size(0),
           temperature);
  } else {
    SoftmaxKernel<thread_bits, R>
          <<<tiles, within_tile, 0, stream>>>
          (mshadow::expr::MakePlan(dst),
           mshadow::expr::MakePlan(src),
           dst.size(0),
           temperature);
  }
}

int main() {
    dali_init();
    TensorInternal<R, 2> bob(mshadow::Shape2(5000, 5000));
    TensorInternal<R, 2> bob_col_softmax(mshadow::Shape2(5000, 5000));

    // set the computing streams
    softmax(bob_col_softmax.mutable_gpu_data(), bob.gpu_data());
    TensorOps::softmax(bob_col_softmax.mutable_gpu_data(), bob.gpu_data());

    int iter = 10;

    for (int i = 0; i < iter; i++) {
        {
            utils::Timer t1("Softmax col-wise (Dali)");
            // our softmax
            softmax(bob_col_softmax.mutable_gpu_data(), bob.gpu_data());
            hipDeviceSynchronize();
        }
        {
            utils::Timer t2("Softmax col-wise (Thrust)");
            // thrust softmax
            TensorOps::softmax(bob_col_softmax.mutable_gpu_data(), bob.gpu_data());
            hipDeviceSynchronize();
        }
        {
            utils::Timer t2("Softmax row-wise (mshadow)");
            // thrust softmax
            TensorOps::softmax_transpose(bob_col_softmax.mutable_gpu_data(), bob.gpu_data());
            hipDeviceSynchronize();
        }
    }

    utils::Timer::report();
}
