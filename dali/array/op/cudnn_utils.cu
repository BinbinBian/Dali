#include "cudnn_utils.h"

#include <atomic>

///////////////////////////////////////////////////////////////////////////////
//                              UTILS                                        //
///////////////////////////////////////////////////////////////////////////////

template<typename T>
struct TensorWrapperApi {
};

template<>
struct TensorWrapperApi<hipdnnTensorDescriptor_t> {
    static void create(hipdnnTensorDescriptor_t* doodle) {
        hipdnnCreateTensorDescriptor(doodle);
    }

    static void destroy(hipdnnTensorDescriptor_t doodle) {
        hipdnnDestroyTensorDescriptor(doodle);
    }

    static void set(hipdnnTensorDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int shape1,
                    int shape2,
                    int shape3,
                    int shape4) {
        hipdnnSetTensor4dDescriptor(
            desc,
            tensor_format,
            dtype,
            shape1,
            shape2,
            shape3,
            shape4);
    }
};

template<>
struct TensorWrapperApi<hipdnnFilterDescriptor_t> {
    static void create(hipdnnFilterDescriptor_t* doodle) {
        hipdnnCreateFilterDescriptor(doodle);
    }

    static void destroy(hipdnnFilterDescriptor_t doodle) {
        hipdnnDestroyFilterDescriptor(doodle);
    }

    static void set(hipdnnFilterDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int shape1,
                    int shape2,
                    int shape3,
                    int shape4) {
        hipdnnSetFilter4dDescriptor(
            desc,
            dtype,
            tensor_format,
            shape1,
            shape2,
            shape3,
            shape4);
    }
};

static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);

// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

namespace cudnn {
    ///////////////////////////////////////////////////////////////////////////
    //                              wrappers                                 //
    ///////////////////////////////////////////////////////////////////////////


    namespace wrapper {
        template<typename Descriptor>
        template<typename T>
        BaseTensor<Descriptor>::BaseTensor(
                TypedArray<memory::DEVICE_T_GPU,T> tensor,
                std::string data_format,
                memory::AM access_mode) {
            hipdnnTensorFormat_t data_format_cudnn;
            if (data_format == "NCHW") {
                data_format_cudnn = HIPDNN_TENSOR_NCHW;
            } else if (data_format == "NHWC") {
                data_format_cudnn = HIPDNN_TENSOR_NHWC;
            }
            hipdnnDataType_t cudnn_dtype;
            if (template_to_dtype<T>() == DTYPE_FLOAT) {
                cudnn_dtype = HIPDNN_DATA_FLOAT;
            } else if (template_to_dtype<T>() == DTYPE_DOUBLE) {
                cudnn_dtype = HIPDNN_DATA_DOUBLE;
            } else {
                ASSERT2(false, "unsupported dtype");
            }
            TensorWrapperApi<Descriptor>::create(&description);
            TensorWrapperApi<Descriptor>::set(
                description,
                data_format_cudnn,
                cudnn_dtype,
                tensor.array.shape()[0],
                tensor.array.shape()[1],
                tensor.array.shape()[2],
                tensor.array.shape()[3]
            );
            // TODO(szymon): add striding support and assert maybe???
            data = tensor.ptr(access_mode);
        }

        template<typename Descriptor>
        BaseTensor<Descriptor>::~BaseTensor() {
            TensorWrapperApi<Descriptor>::destroy(description);
        }

        template class BaseTensor<hipdnnTensorDescriptor_t>;
        template class BaseTensor<hipdnnFilterDescriptor_t>;

        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnTensorDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);

        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
        template BaseTensor<hipdnnFilterDescriptor_t>::BaseTensor(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);




        Convolution::Convolution(int padding_h, int padding_w, int stride_h, int stride_w) {
            hipdnnCreateConvolutionDescriptor(&description);
            hipdnnSetConvolution2dDescriptor(
                description,
                /*pad_h=*/   padding_h,
                /*pad_w=*/   padding_w,
                /*u=*/       stride_h,
                /*v=*/       stride_w,
                /*upscalex=*/1,
                /*upscaley=*/1,
                HIPDNN_CROSS_CORRELATION // Theano issue author claims its twice as fast:
                                        // https://github.com/Theano/Theano/issues/3632
            );
        }

        Convolution::~Convolution() {
            hipdnnDestroyConvolutionDescriptor(description);
        }


        Operator::Operator(OPERATOR_T operator_type, DType dtype) {
            switch (operator_type) {
                case OPERATOR_T_EQL:
                    alpha_d = 1.0;
                    beta_d  = 0.0;
                    break;
                case OPERATOR_T_ADD:
                    alpha_d = 1.0;
                    beta_d  = 1.0;
                    break;
                case OPERATOR_T_SUB:
                    alpha_d = -1.0;
                    beta_d  = 1.0;
                    break;
                default:
                    ASSERT2(false, "Cudnn only supports =, + and - operators");
            }

            alpha_f = alpha_d;
            beta_f  = beta_d;

            if (dtype == DTYPE_FLOAT) {
                alpha_ptr = (void*)&alpha_f;
                beta_ptr  = (void*)&beta_f;
            } else if (dtype == DTYPE_DOUBLE) {
                alpha_ptr = (void*)&alpha_d;
                beta_ptr  = (void*)&beta_d;
            } else {
                ASSERT2(false, "Cudnn only supports floating point types");
            }
        }
    }  // namespace wrapper


    ///////////////////////////////////////////////////////////////////////////
    //                              CONVOLUTIONS                             //
    ///////////////////////////////////////////////////////////////////////////

    void cudnn_conv2d(std::shared_ptr<wrapper::Tensor>  out,
                      std::shared_ptr<wrapper::Tensor>  in,
                      std::shared_ptr<wrapper::Filters> filters,
                      std::shared_ptr<wrapper::Convolution> conv,
                      const wrapper::Operator& update_operator) {

        // TODO(szymon): automatically choose best algorithm.
        hipdnnConvolutionFwdAlgo_t algo =
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
        void* working_memory    = NULL;
        int working_memory_size = 0;

        hipdnnConvolutionForward(
            *get_handle(),
            update_operator.alpha_ptr,
            in->description,
            in->data,
            filters->description,
            filters->data,
            conv->description,
            algo,
            working_memory,
            working_memory_size,
            update_operator.beta_ptr,
            out->description,
            out->data
        );
    }

}  // namespace cudnn
