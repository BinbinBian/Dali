#include "cudnn_utils.h"

#include <atomic>

template<typename T>
struct TensorWrapperApi {
};

template<>
struct TensorWrapperApi<hipdnnTensorDescriptor_t> {
    static void create(hipdnnTensorDescriptor_t* doodle) {
        hipdnnCreateTensorDescriptor(doodle);
    }

    static void destroy(hipdnnTensorDescriptor_t doodle) {
        hipdnnDestroyTensorDescriptor(doodle);
    }

    static void set(hipdnnTensorDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int shape1,
                    int shape2,
                    int shape3,
                    int shape4) {
        hipdnnSetTensor4dDescriptor(
            desc,
            tensor_format,
            dtype,
            shape1,
            shape2,
            shape3,
            shape4);
    }
};

template<>
struct TensorWrapperApi<hipdnnFilterDescriptor_t> {
    static void create(hipdnnFilterDescriptor_t* doodle) {
        hipdnnCreateFilterDescriptor(doodle);
    }

    static void destroy(hipdnnFilterDescriptor_t doodle) {
        hipdnnDestroyFilterDescriptor(doodle);
    }

    static void set(hipdnnFilterDescriptor_t desc,
                    hipdnnTensorFormat_t     tensor_format,
                    hipdnnDataType_t         dtype,
                    int shape1,
                    int shape2,
                    int shape3,
                    int shape4) {
        hipdnnSetFilter4dDescriptor(
            desc,
            dtype,
            tensor_format,
            shape1,
            shape2,
            shape3,
            shape4);
    }
};


template<typename Descriptor>
template<typename T>
DaliCudnnWrapper<Descriptor>::DaliCudnnWrapper(
        TypedArray<memory::DEVICE_T_GPU,T> tensor,
        std::string data_format,
        memory::AM access_mode) {
    hipdnnTensorFormat_t data_format_cudnn;
    if (data_format == "NCHW") {
        data_format_cudnn = HIPDNN_TENSOR_NCHW;
    } else if (data_format == "NHWC") {
        data_format_cudnn = HIPDNN_TENSOR_NHWC;
    }
    hipdnnDataType_t cudnn_dtype;
    if (template_to_dtype<T>() == DTYPE_FLOAT) {
        cudnn_dtype = HIPDNN_DATA_FLOAT;
    } else if (template_to_dtype<T>() == DTYPE_DOUBLE) {
        cudnn_dtype = HIPDNN_DATA_DOUBLE;
    } else {
        ASSERT2(false, "unsupported dtype");
    }
    TensorWrapperApi<Descriptor>::create(&description);
    TensorWrapperApi<Descriptor>::set(
        description,
        data_format_cudnn,
        cudnn_dtype,
        tensor.array.shape()[0],
        tensor.array.shape()[1],
        tensor.array.shape()[2],
        tensor.array.shape()[3]
    );
    // TODO(szymon): add striding support and assert maybe???
    data = tensor.ptr(access_mode);
}

template<typename Descriptor>
DaliCudnnWrapper<Descriptor>::~DaliCudnnWrapper() {
    TensorWrapperApi<Descriptor>::destroy(description);
}

template class DaliCudnnWrapper<hipdnnTensorDescriptor_t>;
template class DaliCudnnWrapper<hipdnnFilterDescriptor_t>;

template DaliCudnnWrapper<hipdnnTensorDescriptor_t>::DaliCudnnWrapper(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
template DaliCudnnWrapper<hipdnnTensorDescriptor_t>::DaliCudnnWrapper(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);

template DaliCudnnWrapper<hipdnnFilterDescriptor_t>::DaliCudnnWrapper(TypedArray<memory::DEVICE_T_GPU,float>, std::string, memory::AM);
template DaliCudnnWrapper<hipdnnFilterDescriptor_t>::DaliCudnnWrapper(TypedArray<memory::DEVICE_T_GPU,double>, std::string, memory::AM);


static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);

// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

namespace cudnn_utils {
    void cudnn_conv2d(std::shared_ptr<DaliCudnnTensor>  out,
                      std::shared_ptr<DaliCudnnTensor>  in,
                      std::shared_ptr<DaliCudnnFilters> filters,
                      int stride_w,
                      int stride_h,
                      int padding_h,
                      int padding_w,
                      double alpha,
                      double beta,
                      DType dtype) {
        void* alpha_ptr;
        void* beta_ptr;
        float alpha_f = alpha, beta_f = beta;

        if (dtype == DTYPE_FLOAT) {
            alpha_ptr = (void*)&alpha_f;
            beta_ptr  = (void*)&beta_f;
        } else if (dtype == DTYPE_DOUBLE) {
            alpha_ptr = (void*)&alpha;
            beta_ptr  = (void*)&beta;
        } else {
            ASSERT2(false, "unsupported dtype");
        }

        hipdnnConvolutionDescriptor_t conv_desc;
        hipdnnCreateConvolutionDescriptor(&conv_desc);
        hipdnnSetConvolution2dDescriptor(
            conv_desc,
            /*pad_h=*/   padding_h,
            /*pad_w=*/   padding_w,
            /*u=*/       stride_h,
            /*v=*/       stride_w,
            /*upscalex=*/1,
            /*upscaley=*/1,
            HIPDNN_CROSS_CORRELATION // theano people say its fast.
        );

        // TODO(szymon): automatically choose best algorithm.
        hipdnnConvolutionFwdAlgo_t algo =
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
        void* working_memory    = NULL;
        int working_memory_size = 0;

        hipdnnConvolutionForward(
            *get_handle(),
            alpha_ptr,
            in->description,
            in->data,
            filters->description,
            filters->data,
            conv_desc,
            algo,
            working_memory,
            working_memory_size,
            beta_ptr,
            out->description,
            out->data
        );

        hipdnnDestroyConvolutionDescriptor(conv_desc);
    }
};
